#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-multi-layer-nn.cu - Multi Layer Neural Network with CUDA
 *
 * Last updated in 2025 by Matteo Fasulo <mat.fasulo@gmail.com>
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * --------------------------------------------------------------------------
 *
 * Compile with:
 * nvcc cuda-multi-layer-nn.cu -o cuda-multi-layer-nn
 *
 * Run with:
 * ./cuda-multi-layer-nn [N] [K]
 *
 * (N = first layer n° neurons; default 1024)
 * (K = number of layers; default 2)
 *
 ****************************************************************************/

#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define BLKDIM 1024
#define R 3
#define BIAS 0.2

/* Define the NeuralNet struct */
/**
 * Neural network struct.
 *
 * @param x The input layer.
 * @param W The weights.
 * @param y The output layer.
 */
struct NeuralNet {
    float *x;
    float *W;
    float *y;
} __align__(16);

/* Define the allocateNeuralNet function */
/**
 * Allocate memory for the neural network.
 *
 * @param net The neural network to allocate.
 * @param N The number of neurons in the first layer.
 * @param M The size of the output layer.
 */
void allocateNeuralNet(NeuralNet &net, const int N, const int M) {
    size_t size_x = N * sizeof(float);
    size_t size_W = N * R * sizeof(float);
    size_t size_y = M * sizeof(float);
    size_t total_size = size_x + size_W + size_y;
    
    
    // hipMalloc returns memory that is usually aligned to 256 bytes.
    float *base_ptr = nullptr;
    hipMalloc((void **)&base_ptr, total_size);
    
    // Slice up the contiguous allocation.
    net.x = base_ptr;
    net.W = (float *)(((char *)base_ptr) + size_x);
    net.y = (float *)(((char *)base_ptr) + size_x + size_W);
}

/* Free the memory allocated for the neural network */
/**
 * @param net The neural network to free.
 * @return void
 */
void freeNeuralNet(NeuralNet &net) {
    if (net.x) {
        hipFree(net.x);
        net.x = net.W = net.y = nullptr;
    }
}

/* Define the Sigmoid function using the math.h lib*/
/**
 * Sigmoid activation function.
 *
 * @param x Input value.
 * @return The sigmoid of the input value.
 */
__device__ float sigmoid(const float x)
{
    return 1.0f / (1.0f + exp(-x));
}

/* Define the fill function */
/**
 * Fill an array with random values.
 *
 * @param array The array to fill.
 * @param size The size of the array.
 */
void fill(float *array, const size_t size)
{
    for (size_t i = 0; i < size; i++)
    {
        array[i] = ((float)rand() / RAND_MAX);
    }
}

/* Define the compute_layer_size function */
/**
 * Compute the size of a layer in the network.
 *
 * @param N The number of neurons in the first layer.
 * @param t The layer index.
 * @return The size of the layer.
 */
int compute_layer_size(const int N, const int t)
{
    return N - t*(R - 1);
}

/* Define the compute_throughput function */
/**
 * Compute the throughput of the network.
 *
 * @param time The time taken to compute the network.
 * @param N The number of neurons in the first layer.
 * @param K The number of layers.
 * @return The throughput of the network.
 */
double compute_throughput(const double time, const int N, const int K)
{
    int processed_items = 0;
    for (int t = 1; t <= K - 1; t++)
    {
        processed_items += compute_layer_size(N, t);
    }
    return processed_items / time;

}

/* Define the forward_propagation kernel without shared memory */
/**
 * Forward propagation kernel. Each thread computes the output of a single neuron in the output layer.
 *
 * @param NeuralNet The neural network.
 * @param in_size Current input layer size.
 */
__global__ void forward_propagation(
    const NeuralNet net,
    const int in_size
) {

    /* Compute the index of the current thread */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int output_size = in_size - R + 1;


    /* Compute the output for this thread */
    if (i < output_size) {
        float sum = BIAS;
        sum += net.x[i + 0] * net.W[(i * R) + 0] +
               net.x[i + 1] * net.W[(i * R) + 1] +
               net.x[i + 2] * net.W[(i * R) + 2];

        net.y[i] = sigmoid(sum);
    }
}

/* Define the forward_propagation kernel with shared memory */
/**
 * Forward propagation kernel with shared memory. Each thread computes the output of a single neuron in the output layer. Shared memory is used to store the input layer since it is accessed multiple times by adjacent threads.
 *
 * @param NeuralNet The neural network.
 * @param in_size Current input layer size.
 */
__global__ void forward_propagation_shared(
    const NeuralNet net,
    const int in_size
) {

    // Declare shared memory
    __shared__ float shared_x[BLKDIM + R - 1];

    /* Compute the index of the current thread */
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    int local_index = threadIdx.x; 

    // Load the main portion of input into shared memory
    if (global_index < in_size) {
        shared_x[local_index] = net.x[global_index];
    } else {
        shared_x[local_index] = 0.0f;
    }

    // Load the extra (R - 1) elements for the boundaries.
    if (threadIdx.x < R - 1) {
        int load_index = global_index + blockDim.x;
        shared_x[local_index + blockDim.x] = (load_index < in_size) ? net.x[load_index] : 0.0f;
    }

    __syncthreads();
    
    // Compute the output layer size
    int output_size = in_size - R + 1;
    
    // Only threads corresponding to valid output indices compute the result.
    if (global_index < output_size) {
        float sum = BIAS;
        // Manual unrolling since R is known at compile time.
        sum += shared_x[local_index + 0] * net.W[(global_index * R) + 0] +
               shared_x[local_index + 1] * net.W[(global_index * R) + 1] +
               shared_x[local_index + 2] * net.W[(global_index * R) + 2];

        net.y[global_index] = sigmoid(sum);
    }
}

int main(int argc, char *argv[])
{
    float *h_x, *h_W, *h_y, *h_y_shared; // Host memory for x, W, and y
    int N = 1024;  // Number of neurons in the first layer
    int K = 2;     // Number of layers
    int M; // Size of the output layer
    double tstart, tstop, tnoshared, tshared; // Timers
    double throughput, shared_throughput; // Throughput in items per second

    if (argc > 3)
    {
        fprintf(stderr, "Usage: %s [N (default %d)] [K (default %d)]\n",
                argv[0], N, K);
        return EXIT_FAILURE;
    }
    if (argc >= 2)
        N = atoi(argv[1]);
    if (argc == 3)
        K = atoi(argv[2]);

    // Validate the input arguments
    if (K < 2)
    {
        fprintf(stderr, "K must be greater than 1.\n");
        return EXIT_FAILURE;
    }
    if (N < 1)
    {
        fprintf(stderr, "N must be a positive integer.\n");
        return EXIT_FAILURE;
    }

    // Compute the size of the output layer
    M = N - R + 1;

    NeuralNet d_nn;
    allocateNeuralNet(d_nn, N, M);

    // Allocate host memory for x, W
    h_x = (float *)malloc(N * sizeof(float)); fill(h_x, N);
    h_W = (float *)malloc(N * R * sizeof(float)); fill(h_W, N * R);
    
    // Copy x and W to device memory
    hipMemcpy(d_nn.x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_nn.W, h_W, N * R * sizeof(float), hipMemcpyHostToDevice);

    int final_layer_size = compute_layer_size(N, K - 1);

    /**
     ** Forward propagation without shared memory
     **/
    printf("No shared memory:\t");

    // Start the time
    tstart = hpc_gettime();
    
    // Forward propagation without shared memory
    for (int t = 1; t <= K - 1; t++)
    {
        int input_layer_size = compute_layer_size(N, t - 1);
        int output_layer_size = compute_layer_size(N, t);

        // Launch the kernel
        forward_propagation<<<(output_layer_size + BLKDIM - 1)/BLKDIM, BLKDIM>>>(
            d_nn,
            input_layer_size
        );

        if (t < K - 1)
        {
            // Swap the input and output arays if we are not at the last layer
            hipMemcpy(d_nn.x, d_nn.y, output_layer_size * sizeof(float), hipMemcpyDeviceToDevice);
        }
    }
    hipDeviceSynchronize();
    // Stop the time
    tstop = hpc_gettime();
    tnoshared = tstop - tstart;
    printf("%fs\n", tnoshared);

    // Calculate throughput
    throughput = compute_throughput(tnoshared, N, K);
    printf("Throughput: %f items/second\n", throughput);

    // Copy the output layer back to the host
    h_y = (float *)malloc(final_layer_size * sizeof(float)); 
    hipMemcpy(h_y, d_nn.y, final_layer_size * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up host and device allocations.
    freeNeuralNet(d_nn);

    // Reallocate memory for the device
    NeuralNet d_nn_shared;
    allocateNeuralNet(d_nn_shared, N, M);
    
    // Copy x and W to device memory
    hipMemcpy(d_nn_shared.x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_nn_shared.W, h_W, N * R * sizeof(float), hipMemcpyHostToDevice);


    printf("Shared memory:\t\t");

    tstart = hpc_gettime();
    // Forward propagation with shared memory
    for (int t = 1; t <= K - 1; t++)
    {
        int input_layer_size = compute_layer_size(N, t - 1);
        int output_layer_size = compute_layer_size(N, t);
        
        // Launch the kernel
        forward_propagation_shared<<<(output_layer_size + BLKDIM - 1)/BLKDIM, BLKDIM>>>(
            d_nn_shared,
            input_layer_size
        );

        if (t < K - 1)
        {
            // Swap the input and output arays if we are not at the last layer
            hipMemcpy(d_nn_shared.x, d_nn_shared.y, output_layer_size * sizeof(float), hipMemcpyDeviceToDevice);
        }
    }
    hipDeviceSynchronize();
    // Stop the time
    tstop = hpc_gettime();
    tshared = tstop - tstart;
    // Print the time and speedup w.r.t the non-shared memory version
    printf("%fs (%.2fx speedup)\n", tshared, tnoshared / tshared);

    // Calculate throughput
    shared_throughput = compute_throughput(tshared, N, K);
    printf("Throughput: %f items/second\n", shared_throughput);

    // Copy the output layer back to the host
    h_y_shared = (float *)malloc(final_layer_size * sizeof(float)); 
    hipMemcpy(h_y_shared, d_nn_shared.y, final_layer_size * sizeof(float), hipMemcpyDeviceToHost);

    // Check if the results are the same
    for (int i = 0; i < final_layer_size; i++)
    {
        if (fabs(h_y[i] - h_y_shared[i]) > 1e-6)
        {
            fprintf(stderr, "Results do not match at index %d: %f != %f\n", i, h_y[i], h_y_shared[i]);
            return EXIT_FAILURE;
        }
    }
    printf("Results match.\n");

    // Clean up host and device allocations.
    freeNeuralNet(d_nn_shared);
    free(h_x); free(h_W); free(h_y); free(h_y_shared);

    return EXIT_SUCCESS;
}